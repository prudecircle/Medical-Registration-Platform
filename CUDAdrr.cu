#include "hip/hip_runtime.h"
#include "CUDAdrr.cuh"
//#include"CUDA_NCC.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include ""
#include <algorithm> 
#include <iostream>
#include <vector>
#include <string>

#define KERNEL                      __global__
#define HOST                        __host__
#define DEVICE                      __device__
#define HOST_AND_DEVICE             __host__ __device__
#define DEVICE_CONST                __device__ __constant__




// This variable contains the DICOM set
float* d_object3D;

// This variable contains the 2D output from CUDA
float* d_object2D;

//X��ͼƬ
cv::Mat m_srcImg;

unsigned char* c_object2D;



// Constants depending on the DICOM
DEVICE_CONST int d_sizeCT[3];
DEVICE_CONST  float ctPixelSpacing[3];

// Constant depending on image output,����DRR�ĳߴ�
DEVICE_CONST int DRRImageSize[2];

// Constants dependion on the specific DRR
DEVICE_CONST  float d_DRR_Parameters[4 * 12 + 3];


// This variable contains the DICOM loaded as a Texture ( read-only, fast-cached memory)
hipTextureObject_t tex_object3D = 0;

hipStream_t stream1;

__global__ void cal_hist(int* hist, int* range) {
	range[0] = 0;
	range[1] = 0;
	for (int i = 0; i < 256; i++) {
		if (hist[i] != 0) {
			range[0] = i;
			break;
		}
	}
	for (int i = 255; i >= 0; i--) {
		if (hist[i] != 0) {
			range[1] = i;
			break;
		}
	}
}
__global__ void hist_uc(unsigned char* object2D, int* range, int PARAS_NUMS) {
	int total_dx = DRRImageSize[0];
	int total_dz = DRRImageSize[1];

	//Every thread calculates its own id number
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	// This checks if the thread number is bigger than the amount of pixels
	if (idx >= total_dx * total_dz * PARAS_NUMS)
		return;

	float tmp = object2D[idx];

	object2D[idx] = (int)((tmp - range[0]) / (range[1] - range[0]) * 255);

}
__device__ float getPixval(hipTextureObject_t tex_object3D, long int idx, int PARAS_NUMS) {

	//-------------kfq7.14
	int total_dx = DRRImageSize[0];
	int total_dz = DRRImageSize[1];

	int idxdz = idx / total_dx;
	int imgIdx = idx / (total_dx * total_dz);
	int dz, dx;

	float stepInX[3];
	float stepInY[3];
	float corner00[3];
	float SourceWorld[3];
	float threshold = d_DRR_Parameters[50];

	stepInX[0] = d_DRR_Parameters[imgIdx * 3];
	stepInX[1] = d_DRR_Parameters[imgIdx * 3 + 1];
	stepInX[2] = d_DRR_Parameters[imgIdx * 3 + 2];
	stepInY[0] = d_DRR_Parameters[12 + imgIdx * 3];
	stepInY[1] = d_DRR_Parameters[13 + imgIdx * 3];
	stepInY[2] = d_DRR_Parameters[14 + imgIdx * 3];
	corner00[0] = d_DRR_Parameters[24 + imgIdx * 3];
	corner00[1] = d_DRR_Parameters[25 + imgIdx * 3];
	corner00[2] = d_DRR_Parameters[26 + imgIdx * 3];
	SourceWorld[0] = d_DRR_Parameters[36 + imgIdx * 3];
	SourceWorld[1] = d_DRR_Parameters[37 + imgIdx * 3];
	SourceWorld[2] = d_DRR_Parameters[38 + imgIdx * 3];
	dz = idxdz - imgIdx * total_dz;
	dx = idx - idxdz * total_dx;


	//Calculate the spatial position of the pixel
	//drrPixelWorld_0[idx] = *corner00_0 + ((*stepInX_0)*(threadIdx.x)) + ((*stepInY_0)*(blockIdx.x));
	//drrPixelWorld_1[idx] = *corner00_1 + ((*stepInX_1)*(threadIdx.x)) + ((*stepInY_1)*(blockIdx.x));
	//drrPixelWorld_2[idx] = *corner00_2 + ((*stepInX_2)*(threadIdx.x)) + ((*stepInY_2)*(blockIdx.x));
	float drrPixelWorld[3] = { 0 };
	drrPixelWorld[0] = corner00[0] + ((stepInX[0]) * dx) + ((stepInY[0]) * dz);
	drrPixelWorld[1] = corner00[1] + ((stepInX[1]) * dx) + ((stepInY[1]) * dz);
	drrPixelWorld[2] = corner00[2] + ((stepInX[2]) * dx) + ((stepInY[2]) * dz);

	//Calculate the ray vector
	float rayVector[3] = { 0 };
	rayVector[0] = drrPixelWorld[0] - SourceWorld[0];
	rayVector[1] = drrPixelWorld[1] - SourceWorld[1];
	rayVector[2] = drrPixelWorld[2] - SourceWorld[2];

	float alpha1[3];
	float alphaN[3];
	float auxalphaMin[3] = { -2, -2, -2 };
	float auxalphaMax[3] = { 2 , 2 , 2 };


	//Calculate alphaMin and alphaMax 
	if (rayVector[2] != 0)
	{
		alpha1[0] = (0.0 - (SourceWorld[2])) / rayVector[2];
		alphaN[0] = ((d_sizeCT[2]) * (ctPixelSpacing[2]) - (SourceWorld[2])) / rayVector[2];
		auxalphaMin[0] = alphaN[0];
		auxalphaMax[0] = alpha1[0];

		if (alpha1[0] < alphaN[0])
		{
			auxalphaMin[0] = alpha1[0];
			auxalphaMax[0] = alphaN[0];
		}
	}

	if (rayVector[1] != 0)
	{
		alpha1[1] = (0.0 - (SourceWorld[1])) / rayVector[1];
		alphaN[1] = ((d_sizeCT[1]) * (ctPixelSpacing[1]) - (SourceWorld[1])) / rayVector[1];
		auxalphaMin[1] = alphaN[1];
		auxalphaMax[1] = alpha1[1];

		if (alpha1[1] < alphaN[1])
		{
			auxalphaMin[1] = alpha1[1];
			auxalphaMax[1] = alphaN[1];
		}
	}


	if (rayVector[0] != 0)
	{
		alpha1[2] = (0.0 - (SourceWorld[0])) / rayVector[0];
		alphaN[2] = ((d_sizeCT[0]) * (ctPixelSpacing[0]) - (SourceWorld[0])) / rayVector[0];
		auxalphaMin[2] = alphaN[2];
		auxalphaMax[2] = alpha1[2];

		if (alpha1[2] < alphaN[2])
		{
			auxalphaMin[2] = alpha1[2];
			auxalphaMax[2] = alphaN[2];
		}
	}


	float alphaMin;

	if (auxalphaMin[0] > auxalphaMin[1]) //x > y
	{
		alphaMin = auxalphaMin[2];
		if (auxalphaMin[0] > alphaMin) { //x > y, x > z
			alphaMin = auxalphaMin[0];
		}
	}
	else //y > x
	{
		alphaMin = auxalphaMin[2];
		if (auxalphaMin[1] > alphaMin)  //y > x, y > z
			alphaMin = auxalphaMin[1];
	}

	float alphaMax;

	if (auxalphaMax[0] < auxalphaMax[1])  // x < y
	{
		alphaMax = auxalphaMax[2];
		if (auxalphaMax[0] < alphaMax)  // x < y, x < z
			alphaMax = auxalphaMax[0];
	}
	else // y < x
	{
		alphaMax = auxalphaMax[2];
		if (auxalphaMax[1] < alphaMax)  // y < x, y < z
			alphaMax = auxalphaMax[1];
	}

	float firstIntersection[3], firstIntersectionIndex[3], firstIntersectionIndexUp[3], firstIntersectionIndexDown[3];

	//Calculate the first intersection of the ray with the planes (alphaX, alphaY and alphaZ)
	firstIntersection[0] = (SourceWorld[0]) + (alphaMin * rayVector[0]);
	firstIntersection[1] = (SourceWorld[1]) + (alphaMin * rayVector[1]);
	firstIntersection[2] = (SourceWorld[2]) + (alphaMin * rayVector[2]);

	firstIntersectionIndex[0] = firstIntersection[0] / (ctPixelSpacing[0]);
	firstIntersectionIndex[1] = firstIntersection[1] / (ctPixelSpacing[1]);
	firstIntersectionIndex[2] = firstIntersection[2] / (ctPixelSpacing[2]);


	firstIntersectionIndexUp[0] = (int)ceil(firstIntersectionIndex[0]);
	firstIntersectionIndexUp[1] = (int)ceil(firstIntersectionIndex[1]);
	firstIntersectionIndexUp[2] = (int)ceil(firstIntersectionIndex[2]);

	firstIntersectionIndexDown[0] = (int)floor(firstIntersectionIndex[0]);
	firstIntersectionIndexDown[1] = (int)floor(firstIntersectionIndex[1]);
	firstIntersectionIndexDown[2] = (int)floor(firstIntersectionIndex[2]);

	float alpha[3] = { 2,2,2 }, alphaIntersectionUp[3], alphaIntersectionDown[3];

	if (rayVector[2] != 0)
	{
		alphaIntersectionUp[2] = (firstIntersectionIndexUp[2] * (ctPixelSpacing[2]) - (SourceWorld[2])) / rayVector[2];
		alphaIntersectionDown[2] = (firstIntersectionIndexDown[2] * (ctPixelSpacing[2]) - (SourceWorld[2])) / rayVector[2];
		alpha[0] = alphaIntersectionDown[2];
		if (alphaIntersectionUp[2] > alpha[0])
			alpha[0] = alphaIntersectionUp[2];
	}

	if (rayVector[1] != 0)
	{
		alphaIntersectionUp[1] = (firstIntersectionIndexUp[1] * (ctPixelSpacing[1]) - (SourceWorld[1])) / rayVector[1];
		alphaIntersectionDown[1] = (firstIntersectionIndexDown[1] * (ctPixelSpacing[1]) - (SourceWorld[1])) / rayVector[1];
		alpha[1] = alphaIntersectionDown[1];
		if (alphaIntersectionUp[1] > alpha[1])
			alpha[1] = alphaIntersectionUp[1];
	}

	if (rayVector[0] != 0)
	{
		alphaIntersectionUp[0] = (firstIntersectionIndexUp[0] * (ctPixelSpacing[0]) - (SourceWorld[0])) / rayVector[0];
		alphaIntersectionDown[0] = (firstIntersectionIndexDown[0] * (ctPixelSpacing[0]) - (SourceWorld[0])) / rayVector[0];
		alpha[2] = alphaIntersectionDown[0];
		if (alphaIntersectionUp[0] > alpha[2])
			alpha[2] = alphaIntersectionUp[0];
	}

	float alphaU[3] = { 999,999,999 };
	//Calculate incremental values (alphaUx, alphaUx, alphaUz) when the ray intercepts the planes
	if (rayVector[2] != 0)
		alphaU[0] = (ctPixelSpacing[2]) / fabs(rayVector[2]);

	if (rayVector[1] != 0)
		alphaU[1] = (ctPixelSpacing[1]) / fabs(rayVector[1]);

	if (rayVector[0] != 0)
		alphaU[2] = (ctPixelSpacing[0]) / fabs(rayVector[0]);


	float U[3] = { -1,-1,-1 };
	// Calculate voxel index incremental values along the ray path
	if ((SourceWorld[2]) < drrPixelWorld[2])
		U[0] = 1;

	if ((SourceWorld[1]) < drrPixelWorld[1])
		U[1] = 1;

	if ((SourceWorld[0]) < drrPixelWorld[0])
		U[2] = 1;


	//Initialize the weighted sum to zero
	float d12 = 0.0, alphaCmin, alphaCminPrev;

	//Initialize the current ray position (alphaCmin)
	if (alpha[0] < alpha[1]) //x < y
	{
		alphaCmin = alpha[2];
		if (alpha[0] < alphaCmin)  //x < y, x < z
			alphaCmin = alpha[0];
	}
	else //y < x
	{
		alphaCmin = alpha[2];
		if (alpha[1] < alphaCmin)  //y < x, y < z
			alphaCmin = alpha[1];
	}

	// Initialize the current voxel index.
	float cIndexNumber[3] = { firstIntersectionIndexDown[0] , firstIntersectionIndexDown[1] , firstIntersectionIndexDown[2] };

	//The while loop represents when the ray is inside the volume
	while (alphaCmin < alphaMax)
	{
		// Store the current ray position 
		alphaCminPrev = alphaCmin;

		if ((alpha[0] <= alpha[1]) && (alpha[0] <= alpha[2])) //Ray front intercepts with x-plane. Update alphaX
		{
			alphaCmin = alpha[0];
			cIndexNumber[2] = cIndexNumber[2] + U[0];
			alpha[0] = alpha[0] + alphaU[0];
		}
		else if ((alpha[1] <= alpha[0]) && (alpha[1] <= alpha[2])) //Ray front intercepts with y-plane. Update alphaY
		{
			alphaCmin = alpha[1];
			cIndexNumber[1] = cIndexNumber[1] + U[1];
			alpha[1] = alpha[1] + alphaU[1];
		}
		else                                                                //Ray front intercepts with z-plane. Update alphaZ
		{
			alphaCmin = alpha[2];
			cIndexNumber[0] = cIndexNumber[0] + U[2];
			alpha[2] = alpha[2] + alphaU[2];
		}


		if ((cIndexNumber[0] >= 0) && (cIndexNumber[0] < (d_sizeCT[0])) &&
			(cIndexNumber[1] >= 0) && (cIndexNumber[1] < (d_sizeCT[1])) &&
			(cIndexNumber[2] >= 0) && (cIndexNumber[2] < (d_sizeCT[2])))
		{
			//If it is a valid index, get the voxel intensity

			int cIndexCoordinate[3] = { static_cast<int> (cIndexNumber[2]) ,static_cast<int> (cIndexNumber[1]) ,static_cast<int> (cIndexNumber[0]) };

			//Get current position from flat object
			long int currentPos3D = cIndexCoordinate[0] + (cIndexCoordinate[1] * (d_sizeCT[2])) + (cIndexCoordinate[2] * (d_sizeCT[2]) * (d_sizeCT[1]));

			//Retrieve intensity value from flat object
			float value = tex1Dfetch<float>(tex_object3D, currentPos3D);

			//Ignore voxels whose intensities are below the desired threshold
			if (value > threshold)
				d12 += value * (alphaCmin - alphaCminPrev);//weighted sum				
		}
	} //end of the while-loop

	float pixval = 255.0 - d12;
	if (pixval < 0)
		pixval = 255.;

	if (pixval > 255)
		pixval = 0.;

	return pixval;
}

__global__ void drrCUDA_uc(unsigned char* object2D, hipTextureObject_t tex_object3D, int* hist, int PARAS_NUMS) {

	int total_dx = DRRImageSize[0];
	int total_dz = DRRImageSize[1];

	//Every thread calculates its own id number
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	/*if (idx == 1) {
		for (int i = 0; i < 39; i++) {
			printf("d_drr[%d]=%d\n", i, d_DRR_Parameters[i]);
		}
	}*/
	// This checks if the thread number is bigger than the amount of pixels
	if (idx >= total_dx * total_dz * PARAS_NUMS)
		return;
	//--------------------kfq 7.12



	/*float pixval1 = getPixval(tex_object3D, idx, 0);
	int tmp1 = (int)pixval1;
	atomicAdd(hist + tmp1, 1);
	object2D1[idx] = (int)pixval1;

	float pixval2 = getPixval(tex_object3D, idx, 1);
	int tmp2 = (int)pixval2;
	atomicAdd(hist + tmp2, 1);
	object2D2[idx] = (int)pixval2;

	float pixval3 = getPixval(tex_object3D, idx, 2);
	int tmp3 = (int)pixval3;
	atomicAdd(hist + tmp3, 1);
	object2D3[idx] = (int)pixval3;*/







	float pixval = getPixval(tex_object3D, idx, PARAS_NUMS);
	int tmp = (int)pixval;
	atomicAdd(hist + tmp, 1);
	/*if (idx < total_dx * total_dz) {
		atomicAdd(hist1 + tmp, 1);
	}
	if (idx >= total_dx * total_dz && idx < 2 * total_dx * total_dz) {
		atomicAdd(hist2 + tmp, 1);
	}
	if (idx >= 2*total_dx * total_dz && idx < 3 * total_dx * total_dz) {
		atomicAdd(hist3 + tmp, 1);
	}
	if (idx >=3* total_dx * total_dz && idx < 4 * total_dx * total_dz) {
		atomicAdd(hist4 + tmp, 1);
	}*/
	//Assign the calculated value for the pixel to its corresponding position in the output array
	object2D[idx] = (int)pixval;
}

void loadDICOMInGPUMemory(float* cpp_object3D, int* sizeCT, float* pixelSpacing)
{
	long int object3Dsize = sizeCT[0] * sizeCT[1] * sizeCT[2];
	hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);


	hipMalloc((void**)&d_object3D, object3Dsize * sizeof(float));
	hipMemcpyAsync(d_object3D, cpp_object3D, object3Dsize * sizeof(float), hipMemcpyHostToDevice, stream1);

	hipMemcpyToSymbol(HIP_SYMBOL(ctPixelSpacing), pixelSpacing, 3 * sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sizeCT), sizeCT, 3 * sizeof(int), 0, hipMemcpyHostToDevice);


	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = d_object3D;
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = object3Dsize * sizeof(float);

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	// create CUDA texture object
	hipDestroyTextureObject(tex_object3D);
	hipCreateTextureObject(&tex_object3D, &resDesc, &texDesc, NULL);

	hipStreamDestroy(stream1);

}
void loadOuputVariablesInGPUMemory(int dimX, int dimZ, int PARAS_NUMS, int flag)
{
	/*long int vectorSize = dimX * dimZ;
	int OutputImageSize[2] = { dimX, dimZ };*/

	//----------------------kfq7.9
	long int vectorSize = dimX * dimZ;
	int OutputImageSize[2] = { dimX, dimZ };

	if (flag == 0)
		hipMalloc((void**)&d_object2D, PARAS_NUMS * vectorSize * sizeof(float));
	else {
		hipMalloc((void**)&c_object2D, PARAS_NUMS * vectorSize * sizeof(unsigned char));


	}

	/*
	DRRImagesize:����DRR�ĳߴ�
	*/

	hipMemcpyToSymbol(HIP_SYMBOL(DRRImageSize), OutputImageSize, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
}

void loadXRayImg(std::string filename)
{
	m_srcImg = cv::imread(filename, cv::IMREAD_GRAYSCALE);
}

void freeDICOMFromGPUMemory()
{
	hipFree(d_object3D);
}

void freeAuxiliaryVariablesInGPUMemory(int flag)
{
	if (flag == 0)
		hipFree(d_object2D);
	else {
		hipFree(c_object2D);


	}
}

void calUCharDRRwithCUDA(CUDAParamerters CUDA_Parameters, DRRParameters DRR_Parameters, int PARAS_NUMS,bool isFrontal)
{
	clock_t start, end, tstart, tend;
	start = clock();
	// size error
	//for (int i = 0; i < 9; i++) {
	//	//std::cout << "stepInX:[" << i << "]= " << DRR_Parameters.stepInX[i] << std::endl;
	//	//std::cout << "stepInY:" << DRR_Parameters.stepInY[i] << std::endl;
	//	std::cout << "corner00:" << DRR_Parameters.corner00[i] << std::endl;
	//	std::cout << "SourceWorld:" << DRR_Parameters.SourceWorld[i] << std::endl;

	//}
	hipMemcpyToSymbol(HIP_SYMBOL(d_DRR_Parameters), DRR_Parameters.stepInX, 51 * sizeof(float), 0, hipMemcpyHostToDevice);

	//Block 6
	int num_Threads = CUDA_Parameters.numThreads;
	int num_Blocks = CUDA_Parameters.numBlocks;

	//------------------------------------------------------------
	//Launching the threads
	int cpu_hist[256] = { 0 };

	int* gpu_hist;

	hipMalloc((int**)&gpu_hist, 64 * PARAS_NUMS * sizeof(int));

	hipMemcpy(gpu_hist, cpu_hist, 64 * PARAS_NUMS * sizeof(int), hipMemcpyHostToDevice);

	drrCUDA_uc << < num_Blocks, num_Threads >> > (c_object2D, tex_object3D, gpu_hist, PARAS_NUMS);

	hipMemcpy(cpu_hist, gpu_hist, 64 * PARAS_NUMS * sizeof(int), hipMemcpyDeviceToHost);

	int* gpu_range;

	hipMalloc((int**)&gpu_range, 2 * sizeof(int));


	cal_hist << <1, 1 >> > (gpu_hist, gpu_range);

	hist_uc << <num_Blocks, num_Threads >> > (c_object2D, gpu_range,PARAS_NUMS);
	//------------------------------------------------------------
	hipFree(gpu_hist);

	hipFree(gpu_range);

	//Copying the result from the calculations from device to host

	hipDeviceSynchronize();
	end = clock();
	//std::cout << "time of drr is:" << (double)(end - start) / CLOCKS_PER_SEC << std::endl;

	long int vectorSize = (int)DRR_Parameters.size[0] * (int)DRR_Parameters.size[1] * PARAS_NUMS;
	
	unsigned char* h_object2D = (unsigned char*)malloc(sizeof(unsigned char) * vectorSize);

	//
	hipMemcpy(h_object2D, c_object2D, vectorSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	cv::Mat image(1024 * PARAS_NUMS, 1024, CV_8UC1, (void*)h_object2D);
	
	
	if (isFrontal == 1) {
		cv::imwrite("tmp.bmp", image);
	}
	else
	{
		cv::imwrite("tmp_2.bmp", image);
	}
	
	
	//cv::imshow("img", image);
	//cv::waitKey(0);
	//cv::destroyAllWindows();

	//��ȡ�ο�ͼ��
	//MainWindow* main = new MainWindow();
	//cv::Mat m_srcImg = cv::imread(main->getXRayPath(), cv::IMREAD_GRAYSCALE);

	//double* result;// = CUDA_NCC(m_srcImg, c_object2D, PARAS_NUMS);
	free(h_object2D);
	hipFree(c_object2D);
	//return result;
}