#include "hip/hip_runtime.h"
#include "CUDA_NCC.cuh"

//ģ��ͼ
uchar* d_m_src;

//����ͼ
//uchar* d_s_src;

//����ͼ���غ�
__device__ int s_sum[256];

//����ͼ���ؾ�ֵ
__device__ double s_mean[256];

//ģ��ͼ���غ�
__device__ int m_sum[64];

//ģ��ͼ���ؾ�ֵ
__device__ double m_mean[64];

//Э����
__device__ int cov[256];

//ģ��ͼ����
__device__ int m_d[64];

//����ͼ����
__device__ int s_d[256];

//�����ϵ��
__device__ double p[256];
__device__ double p_total[4];


//ͼ���С
__device__ __constant__ int d_Imgsize[2];


__global__ void calTotalPixVal(uchar* sample, uchar* model) {
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	//printf("success%d\n",threadIdx.x);
	int n = 8;
	int sub_height = (int)(d_Imgsize[0] / n);
	int sub_width = (int)(d_Imgsize[1] / n);

	//group_idx:�ֲ�ͼƬ������
	int group_idx = (idx / 131072) * 8 + (idx - (idx / 1024) * 1024) / 128;

	//������ͳ�����غ�
	if (group_idx < 64) {
		atomicAdd(&s_sum[group_idx], (int)sample[idx]);
		atomicAdd(&m_sum[group_idx], (int)model[idx]);

	}
	else
	{
		atomicAdd(&s_sum[group_idx], (int)sample[idx]);

	}


}

__global__ void calMean(int PARAS_NUMS) {
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > PARAS_NUMS * 64) return;

	//�ֿ���������ֵ
	if (idx < 64) {
		s_mean[idx] = s_sum[idx] / (d_Imgsize[0] * d_Imgsize[1] / 64); if (s_mean[idx] == 255) s_mean[idx] = 254;
		m_mean[idx] = m_sum[idx] / (d_Imgsize[0] * d_Imgsize[1] / 64); if (m_mean[idx] == 255) m_mean[idx] = 254;
	}
	else {
		s_mean[idx] = s_sum[idx] / (d_Imgsize[0] * d_Imgsize[1] / 64); if (s_mean[idx] == 255) s_mean[idx] = 254;
	}

}



__global__ void calCovDev(uchar* sample, uchar* model, int PARAS_NUMS) {
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int n = 8;
	int sub_height = (int)(d_Imgsize[0] / n);
	int sub_width = (int)(d_Imgsize[1] / n);

	//group_idx:һ��ͼƬ�оֲ�ͼ�������
	int group_idx = (idx / 131072) * 8 + (idx - (idx / 1024) * 1024) / 128;

	if (idx > d_Imgsize[0] * d_Imgsize[1] * PARAS_NUMS) return;

	if (group_idx < 64) {
		atomicAdd(&cov[group_idx], (int)(sample[idx] - s_mean[group_idx]) * (model[idx] - m_mean[group_idx]));
		atomicAdd(&m_d[group_idx], (int)(model[idx] - m_mean[group_idx]) * (model[idx] - m_mean[group_idx]));
		atomicAdd(&s_d[group_idx], (int)(sample[idx] - s_mean[group_idx]) * (sample[idx] - s_mean[group_idx]));
	}
	else
	{
		atomicAdd(&cov[group_idx], (int)(sample[idx] - s_mean[group_idx]) * (model[idx - (int)(idx / 1048576) * 1048576] - m_mean[group_idx - (group_idx / 64) * 64]));
		atomicAdd(&s_d[group_idx], (int)(sample[idx] - s_mean[group_idx]) * (sample[idx] - s_mean[group_idx]));

	}
}



__global__ void warm(int i) {
	printf("****warm****\n");
	printf("s_sum=%d\n", s_sum[0]);
}

__global__ void calNCC(int PARAS_NUMS) {

	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > PARAS_NUMS * 64) return;

	//ÿ���̼߳�����������ϵ��
	p[idx] = (double)cov[idx] / sqrt((double)s_d[idx] * m_d[idx - (idx / 64) * 64]);

}




__global__ void calTotalNCC(int PARAS_NUMS) {
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	//printf("p1=%f\n", p1[0]);
	if (idx > PARAS_NUMS) return;
	for (int i = 0; i < 64; i++) {
		p_total[idx] = p[idx * 64 + i];
	}

}


double* CUDA_NCC(Mat m_srcImg, uchar* d_s_src, int PARAS_NUMS) {

	int height = m_srcImg.rows;
	int width = m_srcImg.cols;

	//����ͼ��ģ��ͼUCHAR����
	uchar* m_src = m_srcImg.data;


	int n = 8;

	int sub_height = (int)(height / n);
	int sub_width = (int)(width / n);


	//ͼ���С
	int Imgsize[2] = { height,width };
	int total_memsize = height * width * sizeof(uchar);
	int sub_Imgsize = sub_height * sub_width;



	//��ʱ����
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);


	//�����Դ�
	/*hipMalloc((void**)&p_total, 4*sizeof(double));*/
	hipMalloc((void**)&d_m_src, total_memsize);
	/*hipMalloc((void**)&s_sum, 256* sizeof(int));
	hipMalloc((void**)&s_mean, 256 * sizeof(double));
	hipMalloc((void**)&m_sum, 64 * sizeof(int));
	hipMalloc((void**)&m_mean, 64 * sizeof(double));
	hipMalloc((void**)&cov, 256 * sizeof(int));
	hipMalloc((void**)&s_d, 256 * sizeof(int));
	hipMalloc((void**)&m_d, 256 * sizeof(int));
	hipMalloc((void**)&p, 256 * sizeof(double));*/

	int int256[256] = { 0 };
	int int64[64] = { 0 };
	double double256[256] = { 0 };
	double double64[64] = { 0 };
	double double4[4] = { 0 };
	hipMemcpyToSymbol(HIP_SYMBOL(s_sum), int256, 256 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(m_sum), int64, 64 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(s_mean), double256, 256 * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(m_mean), double64, 64 * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(cov), int256, 256 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(s_d), int256, 256 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(m_d), int64, 64 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(p), double256, 256 * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(p_total), double4, 4 * sizeof(double), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_Imgsize), Imgsize, 2 * sizeof(int), 0, hipMemcpyHostToDevice);

	hipMemcpy(d_m_src, m_src, total_memsize, hipMemcpyHostToDevice);
	//hipMemcpy(d_s_src, s_src, total_memsize, hipMemcpyHostToDevice);


	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)(height * width * PARAS_NUMS / numThreads);
	//cout << "numBlocks=" << numBlocks << "   numThreads=" << numThreads << endl;

	hipDeviceSynchronize();

	//ͳ�����غ�
	calTotalPixVal << <numBlocks, numThreads >> > (d_s_src, d_m_src);


	//�������ؾ�ֵ
	calMean << <1, 64 * PARAS_NUMS >> > (PARAS_NUMS);


	//���������Э�������
	calCovDev << <numBlocks, numThreads >> > (d_s_src, d_m_src, PARAS_NUMS);


	// hipDeviceSynchronize();
	//��������㻥���ϵ��
	calNCC << <1, 64 * PARAS_NUMS >> > (PARAS_NUMS);

	//ƽ�����������ϵ��
	calTotalNCC << <1, PARAS_NUMS >> > (PARAS_NUMS);


	//hipDeviceSynchronize();
	//int h_p[1] = { 0 };
	//hipMemcpy(h_cov, cov,tile_nums* sizeof(int), hipMemcpyDeviceToHost);


	static double* result = new double[PARAS_NUMS];
	hipMemcpyFromSymbol(result, HIP_SYMBOL(p_total), PARAS_NUMS * sizeof(double), 0, hipMemcpyDeviceToHost);

	hipEventRecord(end, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(end);
	float time;
	hipEventElapsedTime(&time, start, end);
	//cout << "time=" << time / 1000 << endl;
	//cout << "cov=" << h_cov[0] << endl;
	//cout << "p=" << result[0] << endl;


	//cout << h_s_sum[0] << endl;
	//cout << h_m_sum[0] << endl;

	//�ͷ��Դ�

	hipFree(d_m_src);
	/*hipFree(s_sum);
	hipFree(m_sum);
	hipFree(s_mean);
	hipFree(m_mean);
	hipFree(cov);
	hipFree(s_d);
	hipFree(m_d);
	hipFree(p);
	hipFree(p_total);*/

	return result;
}